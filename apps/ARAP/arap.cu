#include "hip/hip_runtime.h"
#include "rxmesh/query.cuh"
#include "rxmesh/rxmesh_static.h"

#include "rxmesh/matrix/sparse_matrix.cuh"

#include "Eigen/Dense"


using namespace rxmesh;

template <typename T>
__device__ __forceinline__ T
edge_cotan_weight(const rxmesh::VertexHandle&       p_id,
                  const rxmesh::VertexHandle&       r_id,
                  const rxmesh::VertexHandle&       q_id,
                  const rxmesh::VertexHandle&       s_id,
                  const rxmesh::VertexAttribute<T>& X)
{
    // Get the edge weight between the two vertices p-r where
    // q and s composes the diamond around p-r

    const vec3<T> p(X(p_id, 0), X(p_id, 1), X(p_id, 2));
    const vec3<T> r(X(r_id, 0), X(r_id, 1), X(r_id, 2));
    const vec3<T> q(X(q_id, 0), X(q_id, 1), X(q_id, 2));
    const vec3<T> s(X(s_id, 0), X(s_id, 1), X(s_id, 2));

    //cotans[(v1, v2)] =np.dot(e1, e2) / np.linalg.norm(np.cross(e1, e2))

    float weight = 0;
    if (q_id.is_valid())
        weight   += dot((p - q), (r - q)) / length(cross(p - q, r - q));
    if (s_id.is_valid())
        weight   += dot((p - s), (r - s)) / length(cross(p - s, r - s));
    weight /= 2;
    return weight;
}



template <typename T, uint32_t blockThreads>
__global__ static void compute_edge_weights(const rxmesh::Context      context,
                                             rxmesh::VertexAttribute<T> coords,
                                            rxmesh::SparseMatrix<T> A_mat)
{

    auto vn_lambda = [&](VertexHandle vertex_id, VertexIterator& vv)
    {
        VertexHandle q_id = vv.back();

        for (uint32_t v = 0; v < vv.size(); ++v) 
        {
            VertexHandle r_id = vv[v];
            T e_weight = 0;
            VertexHandle s_id = (v == vv.size() - 1) ? vv[0] : vv[v + 1];
            e_weight = edge_cotan_weight(vertex_id, r_id, q_id, s_id, coords);
            A_mat(vertex_id, vv[v]) = e_weight;
        }

    };

    auto block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, vn_lambda);
}

template <typename T, uint32_t blockThreads>
__global__ static void compute_edge_weights_evd(const rxmesh::Context      context,
                                            rxmesh::VertexAttribute<T> coords,
                                            rxmesh::SparseMatrix<T>    A_mat)
{

    auto vn_lambda = [&](EdgeHandle edge_id, VertexIterator& vv) {
            T e_weight = 0;
            e_weight = edge_cotan_weight(vv[0], vv[2], vv[1], vv[3], coords);
            A_mat(vv[0], vv[2]) = e_weight;
        
    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::EVDiamond>(block, shrd_alloc, vn_lambda);
}

template <typename T, uint32_t blockThreads>
__global__ static void edge_weight_values(
    const rxmesh::Context      context,
    rxmesh::EdgeAttribute<T> edge_weights,
    rxmesh::SparseMatrix<T>    A_mat)
{

    auto vn_lambda = [&](EdgeHandle edge_id, VertexIterator& ev) {
        edge_weights(edge_id, 0) = A_mat(ev[0], ev[1]);
    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::EV>(block, shrd_alloc, vn_lambda);
}



////////

__host__ __device__ Eigen::Matrix3f calculateSVD(Eigen::Matrix3f S)
{
    Eigen::JacobiSVD<Eigen::Matrix3f, Eigen::ComputeFullU | Eigen::ComputeFullV> svd(S);


    Eigen::MatrixXf V = svd.matrixV();
    //TODO: it should be transpose
    Eigen::MatrixXf U = svd.matrixU().eval(); 

    float smallest_singular_value = svd.singularValues().minCoeff();

    U.col(smallest_singular_value) = U.col(smallest_singular_value) * -1;

    Eigen::MatrixXf R = V * U;

    return R;
}

template <typename T, uint32_t blockThreads>
__global__ static void calculate_rotation_matrix(const rxmesh::Context    context,
                                          rxmesh::VertexAttribute<T> ref_coords,
                                          rxmesh::VertexAttribute<T> current_coords,
                                          rxmesh::VertexAttribute<T>  rotationVector,
                                          rxmesh::SparseMatrix<T> weight_mat)
{

    auto vn_lambda = [&](VertexHandle v_id, VertexIterator& vv) {
        // pi
        
        Eigen::MatrixXf pi = Eigen::MatrixXf::Identity(3, vv.size());
        for (int j = 0; j < vv.size(); j++) 
        {
            pi(0, j) = ref_coords(v_id, 0) - ref_coords(vv[j], 0);
            pi(1, j) = ref_coords(v_id, 1) - ref_coords(vv[j], 1);
            pi(2, j) = ref_coords(v_id, 2) - ref_coords(vv[j], 2);
        }
        
        // Di
        Eigen::VectorXf weight_vector;
        weight_vector.resize(vv.size());

        for (int v = 0; v < vv.size();v++) 
        {
            weight_vector(v) = weight_mat(v_id, vv[v]);
        }
        Eigen::MatrixXf diagonal_mat = weight_vector.asDiagonal();
        
        // pi'T
        Eigen::MatrixXf pi_dash = Eigen::MatrixXf::Identity(3, vv.size());
        for (int j = 0; j < vv.size(); j++) {
            pi_dash(0, j) = current_coords(v_id, 0) - current_coords(vv[j], 0);
            pi_dash(1, j) = current_coords(v_id, 1) - current_coords(vv[j], 1);
            pi_dash(2, j) = current_coords(v_id, 2) - current_coords(vv[j], 2);
        }

        // calculate covariance matrix S = piDiPiTdash
        
        Eigen::Matrix3f S = pi * diagonal_mat * pi_dash.transpose();

        // perform svd on S (eigen)
        
        
        // R =VU


        Eigen::JacobiSVD<Eigen::Matrix3f, Eigen::ComputeFullU | Eigen::ComputeFullV> svd(S);


        /*
        Eigen::MatrixXf V = S.jacobiSvd().matrixV();
        Eigen::MatrixXf U = S.jacobiSvd().matrixU().eval();

        float smallest_singular_value =
            S.jacobiSvd().singularValues().minCoeff();

       U.col(smallest_singular_value)= U.col(smallest_singular_value) * -1;

        Eigen::MatrixXf R = V * U;
        // Matrix R to vector attribute R
        for (int i=0;i<3;i++) {
            for (int j = 0; j < 3; j++)
                rotationVector(v_id, i * 3 + j) = R(i, j);
        }
        */
    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, vn_lambda);
}



/* compute all entries of bMatrix parallely */
template <typename T, uint32_t blockThreads>
__global__ static void calculate_b(const rxmesh::Context    context,
                                          rxmesh::VertexAttribute<T> original_coords, // [num_coord, 3]
                                          rxmesh::DenseMatrix<T>  rot_mat, // [num_coord, 9]
                                          rxmesh::SparseMatrix<T> weight_mat, // [num_coord, num_coord]
                                          rxmesh::MatrixXf<T> bMatrix) // [num_coord, 3]
{
    auto init_lambda = [&](VertexHandle v_id, VertexIterator& vv) {
        // variable to store ith entry of bMatrix
        Eigen::Vector3d bi(0.0f, 0.0f, 0.0f);

        // get rotation matrix for ith vertex
        Eigen::Matrix3f Ri = Eigen::Matrix3f::Zero(3,3)

        for (int i=0;i<3;i++) {
            for (int j = 0; j < 3; j++)
                Ri(i,j) = rot_mat(v_id, i * 3 + j);
        }

        for (int nei_index = 0; nei_index < vv.size();nei_index++) 
        {
            // get weight vector
            Eigen::VectorXf w = weight_mat(v_id, vv[nei_index]); 

            // get rotation matrix for neightbor j
            Eigen::Matrix3f Rj = Eigen::Matrix3f::Zero(3,3)
            for (int i = 0; i < 3; i++) 
                for (int j = 0; j < 3; j++)
                    Rj(i,j) = rot_mat(vv[nei_index], i * 3 + j);
            
            
            Eigen::Matrix3f rot_add = Ri + Rj
            
            // find coord difference
            Eigen::Vector3f vert_diff = original_coords(vid) - original_coords(vv[nei_index])
            
            bi += 0.5 * w * rot_add * vert_diff
        }
        bMatrix[vid] = bi
    };
    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, init_lambda);
}



int main(int argc, char** argv)
{
    Log::init();

    const uint32_t device_id = 0;
    cuda_query(device_id);

    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "sphere3.obj");
    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "dragon.obj");

    //compute wij
    auto weights = rx.add_edge_attribute<float>("edgeWeights", 1);


    auto ref_vertex_pos = *rx.get_input_vertex_coordinates();  // stays same across computation
    auto changed_vertex_pos = rx.add_vertex_attribute<float>("P", 3);  // changes per iteration

    SparseMatrix<float> weight_matrix(rx);

    //obtain cotangent weight matrix
    constexpr uint32_t               CUDABlockSize = 256;
    rxmesh::LaunchBox<CUDABlockSize> launch_box;
    rx.prepare_launch_box({rxmesh::Op::EVDiamond},
                          launch_box,
                          (void*)compute_edge_weights_evd<float, CUDABlockSize>);

     compute_edge_weights_evd<float, CUDABlockSize>
        <<<launch_box.blocks,
                                                  launch_box.num_threads,
                                                  launch_box.smem_bytes_dyn>>>(
                                                  rx.get_context(), ref_vertex_pos, weight_matrix);
    
    //visualise edge weights
     rxmesh::LaunchBox<CUDABlockSize> launch_box2;
     rx.prepare_launch_box(
         {rxmesh::Op::EV},
         launch_box2,
         (void*)edge_weight_values<float, CUDABlockSize>);

     edge_weight_values<float, CUDABlockSize>
         <<<launch_box2.blocks,
            launch_box2.num_threads,
            launch_box2.smem_bytes_dyn>>>(rx.get_context(), *weights, weight_matrix );

     weights->move(DEVICE, HOST);


     //pi and p'i

     //rx.get_polyscope_mesh()->addEdgeScalarQuantity("edgeWeights", *weights);
     //

     //calculate rotation matrix
     auto rot_mat = *rx.add_vertex_attribute<float>("RotationMatrix", 9);

    rxmesh::LaunchBox<CUDABlockSize> rotation_launch_box;

    
    rx.prepare_launch_box({rxmesh::Op::VV},
                           rotation_launch_box,
                           (void*)calculate_rotation_matrix<float, CUDABlockSize>);
    /*
    calculate_rotation_matrix<float, CUDABlockSize>
        <<<rotation_launch_box.blocks,
           rotation_launch_box.num_threads,
           rotation_launch_box.smem_bytes_dyn>>>(rx.get_context(),
                                                 ref_vertex_pos,
                                                 *changed_vertex_pos,
                                                 rot_mat,
                                                 weight_matrix);
                                                 */
                                                 

    



#if USE_POLYSCOPE
    polyscope::show();
#endif
}
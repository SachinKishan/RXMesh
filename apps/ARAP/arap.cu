#include "hip/hip_runtime.h"
#include "rxmesh/query.cuh"
#include "rxmesh/rxmesh_static.h"

#include "rxmesh/matrix/sparse_matrix.cuh"

#include "Eigen/Dense"


using namespace rxmesh;

template <typename T>
__device__ __forceinline__ T
edge_cotan_weight(const rxmesh::VertexHandle&       p_id,
                  const rxmesh::VertexHandle&       r_id,
                  const rxmesh::VertexHandle&       q_id,
                  const rxmesh::VertexHandle&       s_id,
                  const rxmesh::VertexAttribute<T>& X)
{
    // Get the edge weight between the two vertices p-r where
    // q and s composes the diamond around p-r

    const vec3<T> p(X(p_id, 0), X(p_id, 1), X(p_id, 2));
    const vec3<T> r(X(r_id, 0), X(r_id, 1), X(r_id, 2));
    const vec3<T> q(X(q_id, 0), X(q_id, 1), X(q_id, 2));
    const vec3<T> s(X(s_id, 0), X(s_id, 1), X(s_id, 2));

    //cotans[(v1, v2)] =np.dot(e1, e2) / np.linalg.norm(np.cross(e1, e2))

    float weight = 0;
    if (q_id.is_valid())
        weight   += dot((p - q), (r - q)) / length(cross(p - q, r - q));
    if (s_id.is_valid())
        weight   += dot((p - s), (r - s)) / length(cross(p - s, r - s));
    weight /= 2;
    return weight;
}



template <typename T, uint32_t blockThreads>
__global__ static void compute_edge_weights(const rxmesh::Context      context,
                                             rxmesh::VertexAttribute<T> coords,
                                            rxmesh::SparseMatrix<T> A_mat)
{

    auto vn_lambda = [&](VertexHandle vertex_id, VertexIterator& vv)
    {
        VertexHandle q_id = vv.back();

        for (uint32_t v = 0; v < vv.size(); ++v) 
        {
            VertexHandle r_id = vv[v];
            T e_weight = 0;
            VertexHandle s_id = (v == vv.size() - 1) ? vv[0] : vv[v + 1];
            e_weight = edge_cotan_weight(vertex_id, r_id, q_id, s_id, coords);
            A_mat(vertex_id, vv[v]) = e_weight;
        }

    };

    auto block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, vn_lambda);
}

template <typename T, uint32_t blockThreads>
__global__ static void compute_edge_weights_evd(const rxmesh::Context      context,
                                            rxmesh::VertexAttribute<T> coords,
                                            rxmesh::SparseMatrix<T>    A_mat)
{

    auto vn_lambda = [&](EdgeHandle edge_id, VertexIterator& vv) {
            T e_weight = 0;
            e_weight = edge_cotan_weight(vv[0], vv[2], vv[1], vv[3], coords);
            A_mat(vv[0], vv[2]) = e_weight;
        
    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::EVDiamond>(block, shrd_alloc, vn_lambda);
}

template <typename T, uint32_t blockThreads>
__global__ static void edge_weight_values(
    const rxmesh::Context      context,
    rxmesh::EdgeAttribute<T> edge_weights,
    rxmesh::SparseMatrix<T>    A_mat)
{

    auto vn_lambda = [&](EdgeHandle edge_id, VertexIterator& ev) {
        edge_weights(edge_id, 0) = A_mat(ev[0], ev[1]);
    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::EV>(block, shrd_alloc, vn_lambda);
}



////////

__host__ __device__ Eigen::Matrix3f calculateSVD(Eigen::Matrix3f S)
{
    Eigen::JacobiSVD<Eigen::Matrix3f, Eigen::ComputeFullU | Eigen::ComputeFullV> svd(S);


    Eigen::MatrixXf V = svd.matrixV();
    //TODO: it should be transpose
    Eigen::MatrixXf U = svd.matrixU().eval(); 

    float smallest_singular_value = svd.singularValues().minCoeff();

    U.col(smallest_singular_value) = U.col(smallest_singular_value) * -1;

    Eigen::MatrixXf R = V * U;

    return R;
}

template <typename T, uint32_t blockThreads>
__global__ static void calculate_rotation_matrix(const rxmesh::Context    context,
                                          rxmesh::VertexAttribute<T> ref_coords,
                                          rxmesh::VertexAttribute<T> current_coords,
                                          rxmesh::VertexAttribute<T>  rotationVector,
                                          rxmesh::SparseMatrix<T> weight_mat)
{

    auto vn_lambda = [&](VertexHandle v_id, VertexIterator& vv) {
        // pi
        
        Eigen::MatrixXf pi = Eigen::MatrixXf::Identity(3, vv.size());
        for (int j = 0; j < vv.size(); j++) 
        {
            pi(0, j) = ref_coords(v_id, 0) - ref_coords(vv[j], 0);
            pi(1, j) = ref_coords(v_id, 1) - ref_coords(vv[j], 1);
            pi(2, j) = ref_coords(v_id, 2) - ref_coords(vv[j], 2);
        }
        
        // Di
        Eigen::VectorXf weight_vector;
        weight_vector.resize(vv.size());

        for (int v = 0; v < vv.size();v++) 
        {
            weight_vector(v) = weight_mat(v_id, vv[v]);
        }
        Eigen::MatrixXf diagonal_mat = weight_vector.asDiagonal();
        
        // pi'T
        Eigen::MatrixXf pi_dash = Eigen::MatrixXf::Identity(3, vv.size());
        for (int j = 0; j < vv.size(); j++) {
            pi_dash(0, j) = current_coords(v_id, 0) - current_coords(vv[j], 0);
            pi_dash(1, j) = current_coords(v_id, 1) - current_coords(vv[j], 1);
            pi_dash(2, j) = current_coords(v_id, 2) - current_coords(vv[j], 2);
        }

        // calculate covariance matrix S = piDiPiTdash
        
        Eigen::Matrix3f S = pi * diagonal_mat * pi_dash.transpose();

        // perform svd on S (eigen)
        
        
        // R =VU


        Eigen::JacobiSVD<Eigen::Matrix3f, Eigen::ComputeFullU | Eigen::ComputeFullV> svd(S);


        /*
        Eigen::MatrixXf V = S.jacobiSvd().matrixV();
        Eigen::MatrixXf U = S.jacobiSvd().matrixU().eval();

        float smallest_singular_value =
            S.jacobiSvd().singularValues().minCoeff();

       U.col(smallest_singular_value)= U.col(smallest_singular_value) * -1;

        Eigen::MatrixXf R = V * U;
        // Matrix R to vector attribute R
        for (int i=0;i<3;i++) {
            for (int j = 0; j < 3; j++)
                rotationVector(v_id, i * 3 + j) = R(i, j);
        }
        */
    };

    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, vn_lambda);
}



/* compute all entries of bMatrix parallely */
template <typename T, uint32_t blockThreads>
__global__ static void calculate_b(const rxmesh::Context    context,
                                          rxmesh::VertexAttribute<T> original_coords, // [num_coord, 3]
                                          rxmesh::DenseMatrix<T>  rot_mat, // [num_coord, 9]
                                          rxmesh::SparseMatrix<T> weight_mat, // [num_coord, num_coord]
                                          rxmesh::MatrixXf<T> bMatrix) // [num_coord, 3]
{
    auto init_lambda = [&](VertexHandle v_id, VertexIterator& vv) {
        // variable to store ith entry of bMatrix
        Eigen::Vector3d bi(0.0f, 0.0f, 0.0f);

        // get rotation matrix for ith vertex
        Eigen::Matrix3f Ri = Eigen::Matrix3f::Zero(3,3)

        for (int i=0;i<3;i++) {
            for (int j = 0; j < 3; j++)
                Ri(i,j) = rot_mat(v_id, i * 3 + j);
        }

        for (int nei_index = 0; nei_index < vv.size();nei_index++) 
        {
            // get weight vector
            Eigen::VectorXf w = weight_mat(v_id, vv[nei_index]); 

            // get rotation matrix for neightbor j
            Eigen::Matrix3f Rj = Eigen::Matrix3f::Zero(3,3)
            for (int i = 0; i < 3; i++) 
                for (int j = 0; j < 3; j++)
                    Rj(i,j) = rot_mat(vv[nei_index], i * 3 + j);
            
            // find rotation addition 
            Eigen::Matrix3f rot_add = Ri + Rj
            
            // find coord difference
            Eigen::Vector3f vert_diff = original_coords(vid) - original_coords(vv[nei_index])
            
            // update bi
            bi += 0.5 * w * rot_add * vert_diff
        }
        bMatrix[vid] = bi
    };
    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, init_lambda);
}


/* compute system matrix rows parallely (L from eq9) */
template <typename T, uint32_t blockThreads>
__global__ static void calculate_system_matrix(const rxmesh::Context    context,
                                          rxmesh::SparseMatrix<T> weight_mat, // [num_coord, num_coord]
                                          rxmesh::SparseMatrix<T> L) // [num_coord, num_coord]
{
    auto init_lambda = [&](VertexHandle v_id, VertexIterator& vv) {
        for (int nei_index = 0; nei_index < vv.size();nei_index++) 
        {
            L(v_id, v_id) += weight_mat(v_id, vv[nei_index])
            L(v_id, vv[nei_index]) -= weight_mat(v_id, vv[nei_index])
        }

    };
    
    auto                block = cooperative_groups::this_thread_block();
    Query<blockThreads> query(context);
    ShmemAllocator      shrd_alloc;
    query.dispatch<Op::VV>(block, shrd_alloc, init_lambda);
}


int main(int argc, char** argv)
{
    Log::init();

    const uint32_t device_id = 0;
    cuda_query(device_id);

    //RXMeshStatic rx(STRINGIFY(INPUT_DIR) "sphere3.obj");
    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "dragon.obj");

    //compute wij
    auto weights = rx.add_edge_attribute<float>("edgeWeights", 1);


    auto ref_vertex_pos = *rx.get_input_vertex_coordinates();  // stays same across computation
    auto changed_vertex_pos = rx.add_vertex_attribute<float>("P", 3);  // changes per iteration

    SparseMatrix<float> weight_matrix(rx);

    //obtain cotangent weight matrix
    constexpr uint32_t               CUDABlockSize = 256;
    rxmesh::LaunchBox<CUDABlockSize> launch_box;
    rx.prepare_launch_box({rxmesh::Op::EVDiamond},
                          launch_box,
                          (void*)compute_edge_weights_evd<float, CUDABlockSize>);

     compute_edge_weights_evd<float, CUDABlockSize>
        <<<launch_box.blocks,
                                                  launch_box.num_threads,
                                                  launch_box.smem_bytes_dyn>>>(
                                                  rx.get_context(), ref_vertex_pos, weight_matrix);
    
    //visualise edge weights
     rxmesh::LaunchBox<CUDABlockSize> launch_box2;
     rx.prepare_launch_box(
         {rxmesh::Op::EV},
         launch_box2,
         (void*)edge_weight_values<float, CUDABlockSize>);

     edge_weight_values<float, CUDABlockSize>
         <<<launch_box2.blocks,
            launch_box2.num_threads,
            launch_box2.smem_bytes_dyn>>>(rx.get_context(), *weights, weight_matrix );

     weights->move(DEVICE, HOST);


     //pi and p'i

     //rx.get_polyscope_mesh()->addEdgeScalarQuantity("edgeWeights", *weights);
     //

     //calculate rotation matrix
     auto rot_mat = *rx.add_vertex_attribute<float>("RotationMatrix", 9);

    rxmesh::LaunchBox<CUDABlockSize> rotation_launch_box;

    
    rx.prepare_launch_box({rxmesh::Op::VV},
                           rotation_launch_box,
                           (void*)calculate_rotation_matrix<float, CUDABlockSize>);
    /*
    calculate_rotation_matrix<float, CUDABlockSize>
        <<<rotation_launch_box.blocks,
           rotation_launch_box.num_threads,
           rotation_launch_box.smem_bytes_dyn>>>(rx.get_context(),
                                                 ref_vertex_pos,
                                                 *changed_vertex_pos,
                                                 rot_mat,
                                                 weight_matrix);
                                                 */
    
    /*------------------step 2---------------------*/
    /**  Calculate bMatrix */
    uint32_t num_vertices = rx.get_num_vertices();

    
    bMatrix = Eigen::MatrixXd::Zero(num_vertices, 3);

    // call function to calculate bMatrix entries parallely
     rxmesh::LaunchBox<CUDABlockSize> launch_box_bMatrix;
     rx.prepare_launch_box(
         {rxmesh::Op::VV},
         launch_box_bMatrix,
         (void*)calculate_b<float, CUDABlockSize>);

     calculate_b<float, CUDABlockSize>
         <<<launch_box_bMatrix.blocks,
            launch_box_bMatrix.num_threads,
            launch_box_bMatrix.smem_bytes_dyn>>>(rx.get_context(), 
                                                 changed_vertex_pos, 
                                                 rot_mat, 
                                                 weight_matrix, 
                                                 *bMatrix);

    /** Calculate System Matrix L */ 
    systemMatrix = Eigen::MatrixXd::Zero(num_vertices, num_vertices);

    // VertexAttribute that will store 
    auto constraints = *rx.add_vertex_attribute<float>("FixedVertices", 1);

    // call function to calculate L Matrix entries parallely
     rxmesh::LaunchBox<CUDABlockSize> launch_box_L;
     rx.prepare_launch_box(
         {rxmesh::Op::VV},
         launch_box_L,
         (void*)calculate_system_matrix<float, CUDABlockSize>);

     calculate_system_matrix<float, CUDABlockSize>
         <<<launch_box_L.blocks,
            launch_box_L.num_threads,
            launch_box_L.smem_bytes_dyn>>>(rx.get_context(),
                                                 weight_matrix, 
                                                 *systemMatrix);

    // incorporating constraints. Keep the static and user modified vertices the same
    //TODO: check with Ahmed if the following code is correct 
    // (do I need to move matrices from GPU to CPU to run following code?)
    for (int ids:constraints)
    {
        systemMatrix.row(ids).setZero()
        systemMatrix(ids, ids) = 1
    }

    // solve eq9 by Cholesky factorization
    auto coords = rx.get_input_vertex_coordinates();
    std::shared_ptr<DenseMatrix<float>> X_mat = coords->to_matrix();

     // Solving using CHOL
    systemMatrix.pre_solve(PermuteMethod::NSTDIS);
    systemMatrix.solve(bMatrix, *X_mat);

    // move the results to the host
    // if we use LU, the data will be on the host and we should not move the
    // device to the host
    X_mat->move(rxmesh::DEVICE, rxmesh::HOST);

    // copy the results to attributes
    coords->from_matrix(X_mat.get());
    // visualize new position
    rx.get_polyscope_mesh()->updateVertexPositions(*coords);


#if USE_POLYSCOPE
    polyscope::show();
#endif
}
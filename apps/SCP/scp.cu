#include "rxmesh/query.cuh"
#include "rxmesh/rxmesh_static.h"

#include "rxmesh/matrix/sparse_matrix.cuh"

using namespace rxmesh;





int main(int argc, char** argv)
{
    Log::init();

    const uint32_t device_id = 0;
    cuda_query(device_id);

    RXMeshStatic rx(STRINGIFY(INPUT_DIR) "bunnyhead.obj");

    int  number_of_vertices = rx.get_num_vertices();
    auto boundaryVertices = *rx.add_vertex_attribute<int>("boundaryVertices", 1);

//    auto parameter_coords = *rx.add_vertex_attribute();

    rx.get_boundary_vertices(boundaryVertices);

    DenseMatrix<hipComplex> eb(rx, number_of_vertices, 1);
    DenseMatrix<hipComplex> u(rx, number_of_vertices, 1);
    DenseMatrix<hipComplex> T3(rx, number_of_vertices, 1);
    SparseMatrix<hipComplex> B(rx);

    rx.for_each_vertex
    (rxmesh::DEVICE,[B, eb, boundaryVertices] __device__(const rxmesh::VertexHandle vh) mutable
    {
        eb(vh, 0) = make_hipComplex(boundaryVertices(vh, 0) , 0.0f);
        B(vh, vh) = make_hipComplex(boundaryVertices(vh, 0), 0.0f);

    });

    B.move(rxmesh::DEVICE, rxmesh::HOST);
    eb.move(rxmesh::DEVICE, rxmesh::HOST);

    //
    // S = [B- (1/Vb) * ebebT];

    hipComplex T2 = eb.dot(u);
    
    rx.for_each_vertex(
        rxmesh::DEVICE,
        [u, eb, B,T3,T2] __device__(
            const rxmesh::VertexHandle vh) mutable 
        {
            hipComplex T1 = make_hipComplex(B(vh, vh).x * u(vh, 0).x,
                                          B(vh, vh).y * u(vh, 0).y);

            
            T3(vh,0)= hipCsubf(T1,hipCmulf(T2, eb(vh, 0)));

        });

    

    



    rx.get_polyscope_mesh()->addVertexScalarQuantity("vBoundary", boundaryVertices);



#if USE_POLYSCOPE
    polyscope::show();
#endif
}